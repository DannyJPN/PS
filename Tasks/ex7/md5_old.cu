#include "hip/hip_runtime.h"

#include <sys/types.h>
#include <unistd.h>
#include <sys/wait.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <stdio.h>
#include <fcntl.h>
#include <string.h>
#include <stdint.h>
#include <sys/dir.h>
#include <sys/stat.h>
#include <stdio.h>
#include <math.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
using namespace std;
 

// leftrotate function definition
#define LEFTROTATE(x, c) (((x) << (c)) | ((x) >> (32 - (c))))
 
__device__ void to_bytes(uint32_t val, uint8_t *bytes)
{
   bytes[0] = (uint8_t) val;
   bytes[1] = (uint8_t) (val >> 8);
   bytes[2] = (uint8_t) (val >> 16);
   bytes[3] = (uint8_t) (val >> 24);
}
 
__device__ uint32_t to_int32(const uint8_t *bytes)
{
    return (uint32_t) bytes[0]
        | ((uint32_t) bytes[1] << 8)
        | ((uint32_t) bytes[2] << 16)
        | ((uint32_t) bytes[3] << 24);
}
 
__device__ void md5(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest) 
{
 // Constants are the integer part of the sines of integers (in radians) * 2^32.
const uint32_t k[64] = {
0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee ,
0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501 ,
0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be ,
0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821 ,
0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa ,
0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8 ,
0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed ,
0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a ,
0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c ,
0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70 ,
0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05 ,
0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665 ,
0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039 ,
0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1 ,
0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1 ,
0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 };
 
// r specifies the per-round shift amounts
const uint32_t r[] = {7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
                      5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20,
                      4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
                      6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};
 
 
 
 
 
 
 
 
 
 
 
 
 
 
    // These vars will contain the hash
    uint32_t h0, h1, h2, h3;
 
    // Message (to prepare)
    uint8_t *msg = NULL;
 
    size_t new_len, offset;
    uint32_t w[16];
    uint32_t a, b, c, d, i, f, g, temp;
 
    // Initialize variables - simple count in nibbles:
    h0 = 0x67452301;
    h1 = 0xefcdab89;
    h2 = 0x98badcfe;
    h3 = 0x10325476;
 
    //Pre-processing:
    //append "1" bit to message    
    //append "0" bits until message length in bits ≡ 448 (mod 512)
    //append length mod (2^64) to message
 
    for (new_len = initial_len + 1; new_len % (512/8) != 448/8; new_len++)
        ;
 
    msg = (uint8_t*)malloc(new_len + 8);
    memcpy(msg, initial_msg, initial_len);
    msg[initial_len] = 0x80; // append the "1" bit; most significant bit is "first"
    for (offset = initial_len + 1; offset < new_len; offset++)
        msg[offset] = 0; // append "0" bits
 
    // append the len in bits at the end of the buffer.
    to_bytes(initial_len*8, msg + new_len);
    // initial_len>>29 == initial_len*8>>32, but avoids overflow.
    to_bytes(initial_len>>29, msg + new_len + 4);
 
    // Process the message in successive 512-bit chunks:
    //for each 512-bit chunk of message:
    for(offset=0; offset<new_len; offset += (512/8)) {
 
        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        for (i = 0; i < 16; i++)
            w[i] = to_int32(msg + offset + i*4);
 
        // Initialize hash value for this chunk:
        a = h0;
        b = h1;
        c = h2;
        d = h3;
 
        // Main loop:
        for(i = 0; i<64; i++) {
 
            if (i < 16) {
                f = (b & c) | ((~b) & d);
                g = i;
            } else if (i < 32) {
                f = (d & b) | ((~d) & c);
                g = (5*i + 1) % 16;
            } else if (i < 48) {
                f = b ^ c ^ d;
                g = (3*i + 5) % 16;          
            } else {
                f = c ^ (b | (~d));
                g = (7*i) % 16;
            }
 
            temp = d;
            d = c;
            c = b;
            b = b + LEFTROTATE((a + f + k[i] + w[g]), r[i]);
            a = temp;
 
        }
 
        // Add this chunk's hash to result so far:
        h0 += a;
        h1 += b;
        h2 += c;
        h3 += d;
 
    }
 
    // cleanup
    free(msg);
 
    //var char digest[16] := h0 append h1 append h2 append h3 //(Output is in little-endian)
    to_bytes(h0, digest);
    to_bytes(h1, digest + 4);
    to_bytes(h2, digest + 8);
    to_bytes(h3, digest + 12);
}
 


__device__ void WriteArray(int*arr,int arrsize)
{
	for(int i =0;i<arrsize;i++)
	{
		printf("%d",arr[i]);
	}

}


__device__ void WriteArray(char*arr,int arrsize)
{
	for(int i =0;i<arrsize;i++)
	{
		printf("%c",arr[i]);
	}
	
}


__device__ void WriteArrayHex(uint8_t*result,int arrsize)
{
	for (int i = 0; i < arrsize; i++)
	{	
		
		printf("%x", result[i]);
		
	
	}
 printf("\n");
        
	
}


__device__ void gethash(char*msg,size_t len,uint8_t* result)
{
	 
    int i;
    
 

   // len = strlen(msg);
	
    // benchmark
    for (i = 0; i < 500; i++) {
        md5((uint8_t*)msg, len, result);
    }
 
    // display result
	//static char* outbuf=new char[32];
    //static char*outbuf=(char*)malloc(32*sizeof(char));

    //return outbuf;
}






__global__ void GenerateSerie(char *characters,unsigned int charsetlen,const int passlength)
{  

	

 
	unsigned long long totalcount = (unsigned long long)(pow(charsetlen,passlength));
	unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int w = gridDim.x * blockDim.x;
    unsigned int h = gridDim.y * blockDim.y;
    unsigned int threadindex=y*w+x;
	unsigned int threadcount = w*h;
	unsigned long long bindex=0;
	unsigned long long eindex=0;
	unsigned long long singlep=totalcount/threadcount;
    unsigned long long remainder = totalcount%threadcount;
	/*for(unsigned int i = 0;i<threadcount;i++)
    {
        bindex = (i)*singlep;
        eindex = (i+1)*singlep -1;
        if(i == threadcount-1){eindex+=remainder;}
		if(i == threadindex)
		{break;}
    }*/
    
        bindex = (threadindex)*singlep;
        eindex = (threadindex+1)*singlep -1;
        if(threadindex == threadcount-1){eindex+=remainder;}
		 
    
    //printf("Thread %u/%u manages %llu - %llu (%llu ks)\n",threadindex,threadcount,bindex,eindex,eindex-bindex+1);
	  	
		
	const int reslen = 16;
	uint8_t res[reslen];	
	unsigned long long beginpassindex = bindex;
	unsigned long long stoppassindex = eindex;

	if(stoppassindex <= 0 || stoppassindex < beginpassindex)
	{

		stoppassindex =  pow(charsetlen,passlength);
	}
		
		
		
	
	//char result[passlength]={0};
	//unsigned long long indexarray[passlength]={0};
	char * result =(char*)malloc(passlength*sizeof(char));
	unsigned long long* indexarray = (unsigned long long*)malloc(passlength*sizeof(unsigned long long));

	for(int i = 0;i<passlength;i++)
	{
		result[i] = 0;
		indexarray[i] = 0;
	}

	unsigned long long curindex=beginpassindex;

	for(int i=passlength-1;i >= 0;i--)
	{

		unsigned long long in=curindex%charsetlen;
		indexarray[i] = in;
		curindex = curindex/charsetlen;

	}
//printf("Thread %d:B: %llu\n",threadindex,beginpassindex);
//printf("Thread %d:S: %llu\n",threadindex,stoppassindex);


	for(unsigned long long i = beginpassindex;i < stoppassindex;i++)
      {
		
        for(int j = passlength-1;j > 0;j--)
		{
			//printf("%d against %d\n",indexarray[j],charsetlen);
			if(indexarray[j] >= charsetlen )
			{
				
				indexarray[j]=0;
				indexarray[j-1]++;
			}
		}
		
		//WriteArray(indexarray,len);
		//printf("Thread %d with interval %llu - %llu computes string %llu: \n",threadindex,beginpassindex,stoppassindex,i);
		for(int k = 0;k < passlength;k++)
		{

            char c=characters[indexarray[k]];
			result[k]=c;

			
		}
		
		gethash(result,passlength,res);
		//printf("Thread %d with interval %llu - %llu computed that hash of %s is : \t",threadindex,beginpassindex,stoppassindex,result);
		//WriteArray(result,passlength);
		
		
		//WriteArrayHex(res,reslen);
		 
		indexarray[passlength -1]++;
		
		
		
      

		


      }
		
		
		
		
free(result);
free(indexarray);		
	
		
		
		
		
}

void Compute(char *characters,unsigned int charsetlen, int passlength,unsigned int gridDimx,unsigned int gridDimy,unsigned int blockDimx,unsigned int blockDimy)
{

	struct timeval start, end,time_used;
	
	
	
	
	hipError_t cerr;
    unsigned int w = gridDimx * blockDimx;
    unsigned int h = gridDimy * blockDimy;
    unsigned int threadcount = w*h;
	if(threadcount==0)
	{
	printf("Incomputable\n");
	return;
	}
    unsigned long long totalcount = (unsigned long long)(pow(charsetlen,passlength));
    //printf("Blocks[%d,%d] Threads[%d,%d]\n",gridDimx,gridDimy,blockDimx,blockDimy);
	//printf("Total password count %llu\nTotal Threads %u\n",totalcount,threadcount);
	if(totalcount < threadcount)
    {
        //printf("Too many threads for the total amount of passwords. Reducing threads needed.\n");
		while(threadcount>1)
		{
		if(gridDimx >1)gridDimx--;
		w = gridDimx * blockDimx;
		h = gridDimy * blockDimy;
		threadcount = w*h;
		if(totalcount >= threadcount)break;
		
		if(gridDimy >1)gridDimy--;
		w = gridDimx * blockDimx;
		h = gridDimy * blockDimy;
		threadcount = w*h;
		if(totalcount >= threadcount)break;
		
		if(blockDimx >1)blockDimx--;
		w = gridDimx * blockDimx;
		h = gridDimy * blockDimy;
		threadcount = w*h;
		if(totalcount >= threadcount)break;
		
		if(blockDimy >1)blockDimy--;
		w = gridDimx * blockDimx;
		h = gridDimy * blockDimy;
		threadcount = w*h;
		if(totalcount >= threadcount)break;
		
		}
    }
    

    
    //printf("Total of [%d,%d] = %d threads will be used\n",w,h,threadcount);
    //sleep(1);    
    char header[32] = "Threads\tSeconds\n";

	printf("%s",header);
	dim3 grid_size(gridDimx,gridDimy),block_size(blockDimx,blockDimy);

    char* cudachars;
    cerr = hipMalloc( &cudachars, charsetlen * sizeof( char ) );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	
    
   	cerr = hipMemcpy( cudachars, characters, charsetlen * sizeof( char ) , hipMemcpyHostToDevice );
	if ( cerr != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );	

    
    //printf("Generating total of %llu passwords of length %d from %u characters\nThreads %d\n",totalcount,passlength,charsetlen,threadcount);

	gettimeofday(&start, NULL);
	//computation
	GenerateSerie<<< grid_size, block_size >>>(cudachars,charsetlen,passlength);
	
    //computation end
	gettimeofday(&end, NULL);
    
    
    
	timersub(&end, &start, &time_used);
	if ( ( cerr = hipGetLastError() ) != hipSuccess )
	{
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );
	}

	// Output from printf is in GPU memory. 
	// To get its contens it is necessary to synchronize device.


	
	
hipDeviceSynchronize();
	printf("%d\t\t%f",threadcount,(double)time_used.tv_sec + (double)time_used.tv_usec/1000000);	
    
    // Free memory
	hipFree( cudachars );
}